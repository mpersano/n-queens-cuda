
#include <hip/hip_runtime.h>
#include <cassert>
#include <functional>

struct state
{
    int row;
    int left;
    int down;
    int right;
};

__global__ void count_solutions(int size, int *count, state *initial_states, int num_initial_states)
{
    const auto thread_index = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_index >= num_initial_states)
        return;

    constexpr const auto MaxStackSize = 200;
    state stack[MaxStackSize];

    stack[0] = initial_states[thread_index];
    auto stack_top = 1;

    while (stack_top > 0)
    {
        const auto &cur_state = stack[--stack_top];

        const auto row = cur_state.row;
        const auto left = cur_state.left;
        const auto down = cur_state.down;
        const auto right = cur_state.right;

        if (row == size)
        {
            atomicAdd(count, 1);
        }
        else
        {
            const auto used = ~(left | down | right);
            for (auto bit = 1; bit != 1 << size; bit <<= 1)
            {
                if (bit & used)
                {
                    assert(stack_top < MaxStackSize);
                    auto &state = stack[stack_top++];
                    state.row = row + 1;
                    state.left = (left | bit) << 1;
                    state.down = down | bit;
                    state.right = (right | bit) >> 1;
                }
            }
        }
    }
}

int count_solutions_cuda(int size)
{
    constexpr const auto MaxParallelLevel = 5;
    constexpr const auto MaxInitialStates = 20000000;

    state *initial_states;
    hipMallocManaged(&initial_states, MaxInitialStates * sizeof *initial_states);

    int num_initial_states = 0;

    const std::function<void(int, int, int, int, int)> populate_initial_states =
        [initial_states, &num_initial_states, &populate_initial_states](int row, int left, int down, int right, int size)
        {
            assert(row <= MaxParallelLevel);
            const auto used = ~(left | down | right);
            for (auto bit = 1; bit != 1 << size; bit <<= 1)
            {
                if (bit & used)
                {
                    if (row < MaxParallelLevel)
                    {
                        populate_initial_states(row + 1, (left | bit) << 1, down | bit, (right | bit) >> 1, size);
                    }
                    else
                    {
                        assert(num_initial_states < MaxInitialStates);
                        initial_states[num_initial_states++] = {row + 1, (left | bit) << 1, down | bit, (right | bit) >> 1};
                    }
                }
            }
        };

    populate_initial_states(0, 0, 0, 0, size);

    int *count;
    hipMallocManaged(&count, sizeof count);
    *count = 0;

    constexpr const auto ThreadsPerBlock = 256;
    const auto num_blocks = (num_initial_states + ThreadsPerBlock - 1) / ThreadsPerBlock;

    count_solutions<<<num_blocks, ThreadsPerBlock>>>(size, count, initial_states, num_initial_states);
    hipDeviceSynchronize();

    const auto result = *count;

    hipFree(count);
    hipFree(initial_states);

    return result;
}
